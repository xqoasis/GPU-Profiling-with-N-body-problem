#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include<omp.h>
#include "timer.h"
#include <papi.h>

#define BLOCK_SIZE 128
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;
struct timeval timerStart;

void StartTimer(){
  gettimeofday(&timerStart, NULL);
}

double GetTimer(){
  struct timeval timerStop, timerElapsed;
  gettimeofday(&timerStop, NULL);
  timersub(&timerStop, &timerStart, &timerElapsed);

  return timerElapsed.tv_sec*1000.0+timerElapsed.tv_usec/1000.0;
    
}

/*
  initialize in memory as as:
  (particle 0:) x0 y0 z0 0.0 0.0 0.0
  (particle 1:) x1 y1 z1 0.0 0.0 0.0
  ....
*/
void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i+=6){
      for (int j=0;j<=2;++j){	
        data[i+j] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
	data[i+j+3]=0;
      }
  }
}

__global__ void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;   /* p[i].x and p[j].x are generally far apart in memory */
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

void handle_error (int retval){
	printf("PAPI error %d: %s\n", retval, PAPI_strerror(retval));
	exit(1);
}

double my_timer(void){
  struct timeval time;
  gettimeofday(&time, 0);
  return time.tv_sec + time.tv_usec/1000000.0;
}

int main(const int argc, const char** argv) {
  FILE* datafile = NULL;  
  int nBodies;
  int nIters;
  int nt;
  nBodies = atoi(argv[1]);
  nIters  = atoi(argv[2]);
  nt      = atoi(argv[3]);
  const float dt = 0.01f; // time step

  int bytes = nBodies*sizeof(Body);
  float *buf = (float*)malloc(bytes);
  Body *p = (Body*)buf;

  omp_set_num_threads(nt);
  srand(100);
  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Body *d_p = (Body*)d_buf;

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 
  double totalBodyForceTime = 0.0;
  double totalBodyFracForceTime = 0.0;
  int ret;
  double ts, tf, body_ts, body_tf;

  datafile = fopen("nbody.dat","w");  /* open output file */
//  fprintf(datafile,"%d %d %d\n", nBodies, nIters, 0);


  for (int iter = 1; iter <= nIters; iter++) {
    printf("iteration:%d\n", iter);

//    for (int i=0;i<nBodies;++i)
//      fprintf(datafile, "%f %f %f \n", p[i].x, p[i].y, p[i].z);

    StartTimer();

    ts = GetTimer(); 
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);//copy data to GPU
    body_ts = GetTimer();
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces
    body_tf = GetTimer();
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);//copy data back to CPU
    tf = GetTimer();

    #pragma omp parallel for 
    for (int i = 0 ; i < nBodies; i++) { // integrate positions forward
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
      totalBodyForceTime += tf - ts;
      totalBodyFracForceTime += body_tf - body_ts;
    }
    if (iter == 1) {                      
      // First iter is warm up, then start papi
      ret = PAPI_hl_region_begin("gpu1");
      if (ret != PAPI_OK) {
		    handle_error(1);
	    }
    }
  }
  ret = PAPI_hl_region_end("gpu1");
  if (ret != PAPI_OK) {
		handle_error(1);
	}
  fclose(datafile);
  double avgTime = totalTime / (double)(nIters-1); 
  double avgBodyForceTime = totalBodyForceTime / (double)(nIters-1); 
  double avgBodyFracForceTime = totalBodyFracForceTime / (double)(nIters-1);

  printf("avgTime: %f   totTime: %f \n", avgTime, totalTime);
  printf("avgBodyForceTime: %f   totBodyForceTime: %f \n", avgBodyForceTime, totalBodyForceTime);
  printf("avgBodyFracForceTime: %f   totBodyFracForceTime: %f \n", avgBodyFracForceTime, totalBodyFracForceTime);

  free(buf);
  hipFree(d_buf);
}
