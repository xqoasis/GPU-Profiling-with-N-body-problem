#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <papi.h>

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float4 *pos, *vel; } BodySystem;
struct timeval timerStart;

void StartTimer(){
  gettimeofday(&timerStart, NULL);
}

double GetTimer(){
  struct timeval timerStop, timerElapsed;
  gettimeofday(&timerStop, NULL);
  timersub(&timerStop, &timerStart, &timerElapsed);

  return timerElapsed.tv_sec*1000.0+timerElapsed.tv_usec/1000.0;
    
}
void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int tile = 0; tile < gridDim.x; tile++) {
      __shared__ float3 spos[BLOCK_SIZE];
      float4 tpos = p[tile * blockDim.x + threadIdx.x];
      spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
      __syncthreads();

      for (int j = 0; j < BLOCK_SIZE; j++) {
        float dx = spos[j].x - p[i].x;
        float dy = spos[j].y - p[i].y;
        float dz = spos[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
    }

    v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
  }
  
}
void handle_error (int retval){
	printf("PAPI error %d: %s\n", retval, PAPI_strerror(retval));
	exit(1);
}

double my_timer(void){
  struct timeval time;
  gettimeofday(&time, 0);
  return time.tv_sec + time.tv_usec/1000000.0;
}

int main(const int argc, const char** argv) {
  
  int nBodies = 100000;
  if (argc > 1) nBodies = atoi(argv[1]);
  
  const float dt = 0.01f; // time step
  const int nIters = 20;  // simulation iterations
  
  int bytes = 2*nBodies*sizeof(float4);
  float *buf = (float*)malloc(bytes);
  BodySystem p = { (float4*)buf, ((float4*)buf) + nBodies };

  randomizeBodies(buf, 8*nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 
  double totalBodyForceTime = 0.0;
  double totalBodyFracForceTime = 0.0;
  int ret;
  double ts, tf, body_ts, body_tf;

  for (int iter = 1; iter <= nIters; iter++) {
    printf("iteration:%d\n", iter);  	     
    StartTimer();

    ts = GetTimer();
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    body_ts = GetTimer();
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies);
    body_tf = GetTimer();
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
    tf = GetTimer();

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p.pos[i].x += p.vel[i].x*dt;
      p.pos[i].y += p.vel[i].y*dt;
      p.pos[i].z += p.vel[i].z*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
      totalBodyForceTime += tf - ts;
      totalBodyFracForceTime += body_tf - body_ts;
    }
    if (iter == 2) {                      
      // First iter is warm up, then start papi
      ret = PAPI_hl_region_begin("gpu3");
      if (ret != PAPI_OK) {
		    handle_error(1);
	    }
    }

  }
  ret = PAPI_hl_region_end("gpu3");
  if (ret != PAPI_OK) {
		handle_error(1);
	}
  double avgTime = totalTime / (double)(nIters-1); 
  double avgBodyForceTime = totalBodyForceTime / (double)(nIters-1); 
  double avgBodyFracForceTime = totalBodyFracForceTime / (double)(nIters-1); 
  printf("avgTime: %f   totTime: %f \n", avgTime, totalTime);
  printf("avgBodyForceTime: %f   totBodyForceTime: %f \n", avgBodyForceTime, totalBodyForceTime);
  printf("avgBodyFracForceTime: %f   totBodyFracForceTime: %f \n", avgBodyFracForceTime, totalBodyFracForceTime);

  free(buf);
  hipFree(d_buf);
}
